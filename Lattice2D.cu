#include "hip/hip_runtime.h"
//
// Created by maximilian on 30.08.21.
//

#include <random>
#include "Lattice2D.cuh"
#include "InputOutput.h"
#include "global.h"

/**
 * Contains all kernels for lattice-calculation on a 2D Lattice
 * as well as one host method to perform kernel calls
 */
namespace Lattice2D {


    /**
     * Kernels and methods for Langevin-evolution (physical part)
     */
    namespace langevin {

        /**
         * Init lattice with random distribution. Set observables to zero.
         * @param lat
         * @param parameters
         */
        __global__ void device_initLattice(Lattice lat, Parameters parameters) {
            //Check array range
            unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
            if(tid >= parameters.volume) return;

            //Init random states once
            curandStateType randomState{};
            hiprand_init(parameters.seed, tid, 0, &randomState);
            ((curandStateType *)lat.d_curandStateArray)[tid] = randomState;

            //Reset observables
            lat.d_obs_sigma_avg[tid] = 0;
            lat.d_obs_sigma_sq_avg[tid] = 0;
            lat.d_obs_n_avg[tid] = 0;
            lat.d_obs_n_sq_avg[tid] = 0;

            //Init sigma with random values
            lat.d_lattice_sigma[tid] = hiprand_normal(&randomState);
            lat.d_next_lattice_sigma[tid] = hiprand_normal(&randomState);
            lat.d_next_lattice_pi[tid] = hiprand_normal(&randomState);
            lat.d_lattice_pi[tid] = hiprand_normal(&randomState);

            //Create random-vector-field for diffusive random term
            for(int dim = 0; dim < parameters.dimension; dim++) {
                lat.d_random_noise[dim*parameters.volume + tid] = hiprand_normal(&randomState);
                lat.d_next_random_noise[dim*parameters.volume + tid] = hiprand_normal(&randomState);
            }

            //Initialize n with modelA random values
            lat.d_lattice_n[tid] = hiprand_normal(&randomState);
            lat.d_next_lattice_n[tid] = hiprand_normal(&randomState);
            lat.d_lattice_nu[tid] = 0;
            lat.d_next_lattice_nu[tid] = 0;
        }


        /**
         * Use the langevin-equation with the coupled Hamiltonian to calculate the next time-step.
         * A leapfrog-integrator scheme is used.
         * @param time
         * @param lat
         * @param parameters
         */
        __global__ void device_evolveLatticePoint(unsigned int time, Lattice lat, Parameters parameters) {
            //Check array range
            unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
            if(tid >= parameters.volume) return;

            //Calculate indices
            unsigned int x,y,xf1,xf2,xb1,xb2,yf1,yf2,yb1,yb2,xf1yf1,xb1yf1,xf1yb1,xb1yb1;
            {
                y = tid % parameters.sites;
                x = tid / parameters.sites;

                xf1 = parameters.sites * ( (x+1) % parameters.sites) + y;
                xf2 = parameters.sites * ( (x+2) % parameters.sites) + y;
                xb1 = parameters.sites * ( (x-1+parameters.sites) % parameters.sites) + y;
                xb2 = parameters.sites * ( (x-2+parameters.sites) % parameters.sites) + y;

                yf1 = parameters.sites * x + ( (y+1) % parameters.sites);
                yf2 = parameters.sites * x + ( (y+2) % parameters.sites);
                yb1 = parameters.sites * x + ( (y-1+parameters.sites) % parameters.sites);
                yb2 = parameters.sites * x + ( (y-2+parameters.sites) % parameters.sites);

                xf1yf1 = parameters.sites * ( (x+1) % parameters.sites) + ( (y+1) % parameters.sites);
                xf1yb1 = parameters.sites * ( (x+1) % parameters.sites) + ( (y-1+parameters.sites) % parameters.sites);
                xb1yf1 = parameters.sites * ( (x-1+parameters.sites) % parameters.sites) + ( (y+1) % parameters.sites);
                xb1yb1 = parameters.sites * ( (x-1+parameters.sites) % parameters.sites) + ( (y-1+parameters.sites) % parameters.sites);
            }

            //Check for time to apply different parameters for the different stages of thermalization
            if(time == 0) {
                //Leapfrog half step
                parameters.integrator_timeDelta /= 2;
                parameters.integrator_timeDelta_SQRT /= 1.41421f;
            }else if(time < parameters.thermalization_time_1) {
                //External field for thermalization
                parameters.J = parameters.symmetryBreakingJ;
            }else if(parameters.fastThermalization ) {
                if(time-parameters.thermalization_time_1 < parameters.thermalization_time_2/10) {
                    parameters.integrator_timeDelta = 0.1;
                    parameters.integrator_timeDelta_SQRT = 0.3162278;
                }else if(time-parameters.thermalization_time_1 < parameters.thermalization_time_2/5) {
                    parameters.integrator_timeDelta = 0.1;
                    parameters.integrator_timeDelta_SQRT = 0.3162278;
                }
            }


            //Step sigma field (Model A dynamics)
            {
                //step pi
                TFloat next_pi = 0;

                //m^2 * phi
                next_pi -= parameters.msq_A * lat.d_lattice_sigma[tid];

                //nabla^2 phi
                next_pi +=
                        +(lat.d_lattice_sigma[xf1] - 4 * lat.d_lattice_sigma[tid] + lat.d_lattice_sigma[xb1])
                        +(lat.d_lattice_sigma[yf1]  + lat.d_lattice_sigma[yb1]);

                //lambda * phi^3
                next_pi -= parameters.lambda_A/6.0f * POWER_THREE(lat.d_lattice_sigma[tid]);

                //external field: J
                next_pi += parameters.J;

                //Langevin damping
                next_pi -= parameters.gamma_A * lat.d_lattice_pi[tid];

                //Coupling
                next_pi -= parameters.C * lat.d_lattice_n[tid];

                next_pi *= parameters.integrator_timeDelta;

                //Random term
                next_pi += parameters.integrator_timeDelta_SQRT * parameters.standard_deviation_A
                           * hiprand_normal(&((curandStateType *)lat.d_curandStateArray)[tid]);

                next_pi += lat.d_lattice_pi[tid];

                //Update pi,phi values for field
                lat.d_next_lattice_pi[tid] = next_pi;
                lat.d_next_lattice_sigma[tid] = lat.d_lattice_sigma[tid] + parameters.integrator_timeDelta * next_pi;
            }

            //Step n field with model A dynamics
            if(parameters.thermalization_time_3 == 0 || time < int(parameters.thermalization_time_1 + parameters.thermalization_time_2)) {
                //step pi (= nu)
                TFloat next_pi = 0;

                //m^2 * phi
                next_pi -= parameters.msq_B * lat.d_lattice_n[tid];

                //nabla^2 phi
                next_pi +=
                        +(lat.d_lattice_n[xf1] - 4 * lat.d_lattice_n[tid] + lat.d_lattice_n[xb1])
                        +(lat.d_lattice_n[yf1]  + lat.d_lattice_n[yb1]);

                //external field: J
                next_pi += parameters.J;

                next_pi -= parameters.gamma_B * lat.d_lattice_nu[tid];

                next_pi -= parameters.C * lat.d_lattice_sigma[tid];

                next_pi *= parameters.integrator_timeDelta;

                next_pi += parameters.integrator_timeDelta_SQRT * parameters.standard_deviation_A *
                           (hiprand_normal(&((curandStateType *)lat.d_curandStateArray)[tid]));

                next_pi += lat.d_lattice_nu[tid];

                //Update pi,phi values for field
                lat.d_next_lattice_nu[tid] = next_pi;
                lat.d_next_lattice_n[tid] = lat.d_lattice_n[tid] + parameters.integrator_timeDelta * next_pi;
            }

            else
                //Step n field with model B dynamics
            {
                //step nu
                TFloat next_nu = 0;

                //m^2 * nabla^2 n
                next_nu +=
                        parameters.msq_B * (
                                //nabla
                                lat.d_lattice_n[xf1] + lat.d_lattice_n[xb1] + lat.d_lattice_n[yf1] + lat.d_lattice_n[yb1] - 4 * lat.d_lattice_n[tid]
                        );

                next_nu -= 0
                           +       lat.d_lattice_n[xb2] + lat.d_lattice_n[xf2] + lat.d_lattice_n[yb2] + lat.d_lattice_n[yf2]
                           +  2 * (lat.d_lattice_n[xb1yf1]+lat.d_lattice_n[xf1yf1]+lat.d_lattice_n[xb1yb1]+lat.d_lattice_n[xf1yb1])
                           -  8 * (lat.d_lattice_n[xb1]+lat.d_lattice_n[xf1]+lat.d_lattice_n[yb1]+lat.d_lattice_n[yf1])
                           + 20 *  lat.d_lattice_n[tid];

                //Coupling
                next_nu +=
                        parameters.C * ( //nabla
                                +(lat.d_lattice_sigma[xf1] - 4 * lat.d_lattice_sigma[tid] + lat.d_lattice_sigma[xb1])
                                +(lat.d_lattice_sigma[yf1]  + lat.d_lattice_sigma[yb1])
                        );

                // mu * (...)
                next_nu *= parameters.mu;

                //Langevin damping
                next_nu -= parameters.gamma_B * lat.d_lattice_nu[tid];

                //leapfrog
                next_nu *= parameters.integrator_timeDelta;

                //randomNoise with sqrt(integrator_timeDelta)
                next_nu += parameters.integrator_timeDelta_SQRT * parameters.standard_deviation_B * (
                        lat.d_random_noise[tid] + lat.d_random_noise[parameters.volume + tid]
                        - lat.d_random_noise[xb1] - lat.d_random_noise[parameters.volume + yb1]
                );

                //leapfrog
                next_nu += lat.d_lattice_nu[tid];

                //Update pi,phi values for field (leapfrog)
                lat.d_next_lattice_nu[tid] = next_nu;
                lat.d_next_lattice_n[tid] = lat.d_lattice_n[tid] + parameters.integrator_timeDelta * next_nu;
            }

            //Fill next random vector field (for x and y dimension)
            lat.d_next_random_noise[tid] = hiprand_normal(&((curandStateType *)lat.d_curandStateArray)[tid]);
            lat.d_next_random_noise[parameters.volume + tid] = hiprand_normal(&((curandStateType *)lat.d_curandStateArray)[tid]);

            if(parameters.flag_printEveryStep) {
                //Measure observables (overwrite mode)
                lat.d_obs_sigma_avg[tid] = (lat.d_lattice_sigma[tid]);
                lat.d_obs_sigma_sq_avg[tid] = (POWER_TWO(lat.d_lattice_sigma[tid]));
                lat.d_obs_pi_avg[tid] = (lat.d_lattice_pi[tid]);
                lat.d_obs_pi_sq_avg[tid] = (POWER_TWO(lat.d_lattice_pi[tid]));
                lat.d_obs_n_avg[tid] = (lat.d_lattice_n[tid]);
                lat.d_obs_n_sq_avg[tid] = (POWER_TWO(lat.d_lattice_n[tid]));
                lat.d_obs_nu_sq_avg[tid] = (POWER_TWO(lat.d_lattice_nu[tid]));
                lat.d_obs_nu_avg[tid] = (lat.d_lattice_nu[tid]);
            }
            else
            if(time >= parameters.thermalization_time_3+parameters.thermalization_time_2+parameters.thermalization_time_1) {
                //Measure observables (add mode)
                lat.d_obs_sigma_avg[tid] += (lat.d_lattice_sigma[tid]);
                lat.d_obs_n_avg[tid] += (lat.d_lattice_n[tid]);
                lat.d_obs_sigma_sq_avg[tid] += (POWER_TWO(lat.d_lattice_sigma[tid]));
                lat.d_obs_n_sq_avg[tid] += (POWER_TWO(lat.d_lattice_n[tid]));
            }
        }


        /**
         * If the model dynamics are changed, the nu field needs to be reinitialized
         * @param lat
         * @param parameters
         */
        __global__ void device_reinitializeNuField(Lattice lat, Parameters parameters) {
            //Check array range
            unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
            if(tid >= parameters.volume) return;

            //Calculate indexes
            unsigned int y = tid % parameters.sites;
            unsigned int x = tid / parameters.sites;
            unsigned int xb1 = parameters.sites * ( (x-1+parameters.sites) % parameters.sites) + y;

            //Divergence of the random-vector-field -> Diffusive random term
            lat.d_lattice_nu[tid] = sqrt(parameters.temperature) * (
                    lat.d_random_noise[tid] + lat.d_random_noise[parameters.volume + tid]
                    - lat.d_random_noise[xb1] - lat.d_random_noise[parameters.volume + xb1]
            );

            //Fill next random vector field (for x and y dimension)
            lat.d_next_random_noise[tid] = hiprand_normal(&((curandStateType *)lat.d_curandStateArray)[tid]);
            lat.d_next_random_noise[parameters.volume + tid] = hiprand_normal(&((curandStateType *)lat.d_curandStateArray)[tid]);
        }

        /**
         * Subtract the nu-average from every nu-lattice site.
         * @param lattice
         * @param parameters
         */
        void modelChange_subtractNuAverageFromNuField(const Lattice& lattice, const Parameters& parameters) {
            auto * buffer = new TFloat [parameters.volume];
            hipMemcpy(buffer, lattice.d_lattice_nu, parameters.volume * sizeof(TFloat), hipMemcpyDeviceToHost);

            //Use double precision for average calculation
            double nu_avg = 0;
            for(size_t i = 0; i < parameters.volume; i++) {
                nu_avg += (double) buffer[i];
            }
            nu_avg /= (double)parameters.volume;
            for(size_t i = 0; i < parameters.volume; i++) {
                buffer[i] -= (TFloat) nu_avg;
            }

            hipMemcpy(lattice.d_lattice_nu, buffer, parameters.volume * sizeof(TFloat), hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            delete[] buffer;
        }

        /**
         * Set the nu-field to zero
         * @param lattice
         * @param parameters
         */
        void modelChange_setNuFieldZero(const Lattice& lattice, const Parameters& parameters) {
            auto * buffer = new TFloat [parameters.volume];
            //Set local array to zero
            for(size_t i = 0; i < parameters.volume; i++) {buffer[i] = 0;}
            //Push local array on device
            hipMemcpy(lattice.d_lattice_nu, buffer, parameters.volume * sizeof(TFloat), hipMemcpyHostToDevice);
            hipMemcpy(lattice.d_next_lattice_nu, buffer, parameters.volume * sizeof(TFloat), hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            delete[] buffer;
            std::cout << "ZERO SET" << std::endl;
        }

    }

    namespace observables {
        //====== Observables
        __global__ void reduceArray(float * d_array, size_t length, size_t depth) {
            size_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;

            //get indices for summation from tid and depth
            size_t first = (1 << depth) * tid;
            size_t second = (1 << depth) * (tid) + (1 << (depth-1));

            if(second >= length){
                //if only second index exceeds array, average firstIndex with zero
                if(first < length) { d_array[first] /= 2; }
                return;
            }
            //sum array
            d_array[first] += d_array[second];
            //average sum
            d_array[first] /= 2;
        }

        TFloat getFieldAverage(float * d_array, size_t length) {
            //smallest power-2 number greater than length; needed for correcting the value if length is not a power-2 number
            unsigned long long powerLength = 1;
            //loop as long as 2**depth < length
            for(int depth = 1; (1 << depth) < 2*length; depth++) {
                dim3 grid = (length/depth) / NUM_THREADS + 1;
                reduceArray<<<grid,NUM_THREADS>>>(d_array, length,depth);
                hipDeviceSynchronize();
                powerLength <<= 1;
            }
            TFloat arraySum = 0;
            hipMemcpy(&arraySum, d_array, sizeof(TFloat), hipMemcpyDeviceToHost);
            //Correct value if length is not a power of two and return
            return arraySum * (float)((double)powerLength/(double)length);
        }

        /**
         * Get all observables form device using ArrayReduction
         * Caution: This modifies the observables Arrays!
         * TODO copy to buffer array in first step to avoid modification of the org array
         * @param lattice
         * @param parameters
         * @return
         */
        Observables getObservables(const Lattice& lattice, const Parameters& parameters) {
            Observables observables;

            observables.sigma_avg = getFieldAverage(lattice.d_obs_sigma_avg, parameters.volume);
            observables.sigma_sq_avg = getFieldAverage(lattice.d_obs_sigma_sq_avg, parameters.volume);
            observables.pi_avg = getFieldAverage(lattice.d_obs_pi_avg, parameters.volume);
            observables.pi_sq_avg = getFieldAverage(lattice.d_obs_pi_sq_avg, parameters.volume);
            observables.n_avg = getFieldAverage(lattice.d_obs_n_avg, parameters.volume);
            observables.n_sq_avg = getFieldAverage(lattice.d_obs_n_sq_avg, parameters.volume);
            observables.nu_avg = getFieldAverage(lattice.d_obs_nu_avg, parameters.volume);
            observables.nu_sq_avg = getFieldAverage(lattice.d_obs_nu_sq_avg, parameters.volume);

            hipError_t cuda_err = hipDeviceSynchronize();
            if (cuda_err != hipSuccess) printf("hipError_t: \"%s\".\n", hipGetErrorString(cuda_err));

            return observables;
        }

    }


    /**
     * Starts a single langevin simulation with the given parameters and lattice
     * The lattice needs to be allocated. But not initialized
     * The device-lattices are changed
     * @param parameters Parameters for simulation
     * @param lattice Allocated lattice on the device
     * @return
     */
    Observables startSimulation(Parameters parameters, Lattice lattice) {

        //For storing all time steps if parameters.flag_printEveryStep is enabled
        std::map<std::string, std::vector<TFloat>> stepMap;
        {
            stepMap["sigma"] = std::vector<TFloat>();
            stepMap["sigma_sq"] = std::vector<TFloat>();
            stepMap["pi"] = std::vector<TFloat>();
            stepMap["pi_sq"] = std::vector<TFloat>();
            stepMap["n"] = std::vector<TFloat>();
            stepMap["n_sq"] = std::vector<TFloat>();
            stepMap["nu"] = std::vector<TFloat>();
            stepMap["nu_sq"] = std::vector<TFloat>();
        }

        //Number of Cuda-cores to invoke
        dim3 grid = parameters.volume / NUM_THREADS + 1;

        //Init lattice
        langevin::device_initLattice<<<grid,NUM_THREADS>>>(lattice, parameters);
        hipDeviceSynchronize();

        //Set external field for thermalization
        //If programOption is zero choose randomly between +0.1 and - 0.1
        if(parameters.symmetryBreakingJ == 0) {
            std::default_random_engine generator(parameters.seed);
            std::uniform_real_distribution<TFloat> distribution(0,1);
            parameters.symmetryBreakingJ = distribution(generator) < 0.5f ? -1 : 1;
        }

        //Start time evolution
        for(int time = 0 ; time < parameters.thermalization_time_1 + parameters.thermalization_time_2
                                + parameters.thermalization_time_3 + parameters.measure_time; time++) {

            //Thermalization ModelChange correction
            if(parameters.thermalization_time_3 != 0 &&
                time == parameters.thermalization_time_1 + parameters.thermalization_time_2) {
                if(parameters.dynamicChangeMode == DYNAMIC_CHANGE_MODE_RE_INIT) {
                    langevin::device_reinitializeNuField<<<grid, NUM_THREADS>>>(lattice, parameters);
                    hipDeviceSynchronize();
                }else if(parameters.dynamicChangeMode == DYNAMIC_CHANGE_MODE_AVG_SUBTRACT) {
                    langevin::modelChange_subtractNuAverageFromNuField(lattice, parameters);
                }else if(parameters.dynamicChangeMode == DYNAMIC_CHANGE_MODE_ZERO) {
                    langevin::modelChange_setNuFieldZero(lattice,parameters);
                }
            }

            //Update every lattice-site
            langevin::device_evolveLatticePoint<<<grid, NUM_THREADS>>>(time, lattice,parameters);
            hipError_t cuda_err = hipDeviceSynchronize();
            if (cuda_err != hipSuccess) {
                printf("hipError_t: \"%s\".\n", hipGetErrorString(cuda_err));
                std::exit(1);
            }

            //Swap field-pointers for next calculation
            std::swap(lattice.d_lattice_sigma, lattice.d_next_lattice_sigma);
            std::swap(lattice.d_lattice_pi, lattice.d_next_lattice_pi);
            std::swap(lattice.d_lattice_n, lattice.d_next_lattice_n);
            std::swap(lattice.d_lattice_nu, lattice.d_next_lattice_nu);
            std::swap(lattice.d_random_noise, lattice.d_next_random_noise);

            if(parameters.flag_printEveryStep) {
                Observables stepObs = observables::getObservables(lattice,parameters);

                std::cout << "t=" << time <<"\t";
                printObservables(stepObs);

                //Record sigma and n
                stepMap["sigma"].push_back(stepObs.sigma_avg);
                stepMap["sigma_sq"].push_back(stepObs.sigma_sq_avg);
                stepMap["pi"].push_back(stepObs.pi_avg);
                stepMap["pi_sq"].push_back(stepObs.pi_sq_avg);
                stepMap["n"].push_back(stepObs.n_avg);
                stepMap["n_sq"].push_back(stepObs.n_sq_avg);
                stepMap["nu"].push_back(stepObs.nu_avg);
                stepMap["nu_sq"].push_back(stepObs.nu_sq_avg);
            }
        }

        //At this point the simulation is finished

        //Get observables from lattice after measurement
        Observables observables = observables::getObservables(lattice, parameters);

        //Normalize observables
        if(!parameters.flag_printEveryStep) {
            observables.sigma_avg /= (TFloat) parameters.measure_time;
            observables.sigma_sq_avg /= (TFloat) parameters.measure_time;
            observables.n_avg /= (TFloat) parameters.measure_time;
            observables.n_sq_avg /= (TFloat) parameters.measure_time;
        } else {
            writeAllSteps(parameters, stepMap);
        }

        return observables;
    }

}
